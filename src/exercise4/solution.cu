#include "hip/hip_runtime.h"


#include <stdio.h>


#define CHECK_CUDA_ERROR(call) { \
    hipError_t err = call;     \
    if (err != hipSuccess) {   \
        fprintf(stderr, "CUDA error in file '%s' in line %i: %s.\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE);     \
    }                           \
}

__global__ void dynamic_parallelism_check(int size, int depth) {
    printf("Depth %d, bid %d, tid %d\n", depth, blockIdx.x, threadIdx.x);

    if (size == 1)
        return;

    if (blockIdx.x == 0 & threadIdx.x == 0)
        dynamic_parallelism_check<<<2, size / 2>>>(size / 2, depth + 1);
}


int main() {
    
    dynamic_parallelism_check<<<2, 8>>>(8, 0);

    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    CHECK_CUDA_ERROR(hipDeviceReset());
    return 0;
}
